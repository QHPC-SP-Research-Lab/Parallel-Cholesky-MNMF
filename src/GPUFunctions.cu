#include "hip/hip_runtime.h"
#include "GPUdefines.cuh"


__global__ void cFinalTraceForCublas(const unsigned int n, const MyComplex Num, MyComplex *X)
{
  // For only one block with 32 threads where n >= 32 //
  MyComplex ctmp;

  ctmp.x=ctmp.y=0.0;

  for(unsigned int i=0; i<n; i+=32)
    if ((threadIdx.x+i) < n)
      ctmp=hipCadd(ctmp, X[(threadIdx.x+i)*n+threadIdx.x+i]);

  ctmp.x=ReduceWarp(ctmp.x);
  ctmp.y=ReduceWarp(ctmp.y);

  if (threadIdx.x == 0)
    X[0]=hipCdiv(Num, ctmp);
}


__global__ void dFinalTraceForCublas(const unsigned int n, const MyType Num, MyType *X)
{
  // For only one block with 32 threads where n >= 32 //
  MyType dtmp=0.0;
  for(unsigned int i=0; i<n; i+=32)
    if ((threadIdx.x+i) < n)
      dtmp += X[(threadIdx.x+i)*n+threadIdx.x+i];
  dtmp=ReduceWarp(dtmp);
  if (threadIdx.x == 0)
    X[0]=Num/dtmp;
}


__global__ void cFinalTraceForKernel(const unsigned int n, const MyComplex *B, MyComplex *C, MyComplex *R)
{
  // For only one block with 32 threads where n >= 32 //
  MyComplex nume, deno;
  
  nume.x=nume.y=deno.x=deno.y=0.0;
  
  for(unsigned int i=0; i<n; i+=32)
    if ((threadIdx.x+i) < n)
    {
      nume=hipCadd(nume, B[threadIdx.x+i]);
      deno=hipCadd(deno, C[(threadIdx.x+i)*n+threadIdx.x+i]);
    }

  nume.x=ReduceWarp(nume.x); nume.y=ReduceWarp(nume.y); 
  deno.x=ReduceWarp(deno.x); deno.y=ReduceWarp(deno.y);

  if (threadIdx.x == 0)
    R[0]=hipCdiv(nume, deno);
}


__global__ void dFinalTraceForKernel(const unsigned int n, const MyType *B, MyType *C, MyType *R)
{
  // For only one block with 32 threads where n >= 32 //
  MyType nume=0.0, deno=0.0;
  for(unsigned int i=0; i<n; i+=32)
  {
    if ((threadIdx.x+i) < n)
    {
      nume += B[threadIdx.x+i];
      deno += C[(threadIdx.x+i)*n+threadIdx.x+i];
    }
  }
  nume=ReduceWarp(nume);
  deno=ReduceWarp(deno);

  if (threadIdx.x == 0)
    R[0]=nume/deno;
}


__global__ void cTraceHalfCoale(const unsigned int n, MyComplex *A, const MyComplex *B)
{
  if (blockIdx.x < n)
  {
    unsigned int yo=(blockIdx.x * n + threadIdx.x), pos=threadIdx.x*n+blockIdx.x, i;
    MyComplex ctmp;

    ctmp.x=ctmp.y=0.0;

    for(i=0; i<n; i+=32)
    {
      if (threadIdx.x+i < n)
        // hipCfma(x,y,z) --> x*y+z
        ctmp=hipCfma(A[i*n+pos], B[yo+i], ctmp);
    }

    ctmp.x=ReduceWarp(ctmp.x);
    ctmp.y=ReduceWarp(ctmp.y);

    if (threadIdx.x == 0)
      A[blockIdx.x]=ctmp;
  }
}


__global__ void dTraceHalfCoale(const unsigned int n, MyType *A, const MyType *B)
{
  if (blockIdx.x < n)
  {
    unsigned int yo=(blockIdx.x * n + threadIdx.x), pos=threadIdx.x*n+blockIdx.x, i;
    MyType dtmp= 0.0;
    for(i=0; i<n; i+=32)
    {
      if (threadIdx.x+i < n)
        dtmp += A[i*n+pos]*B[yo+i];
    }
    dtmp=ReduceWarp(dtmp);
    if (threadIdx.x == 0) A[blockIdx.x]=dtmp;
  }
}


int ComplexOneClassicStep(hipsolverHandle_t handle, const int NChan, const int SizeTRF, int *Info, MyComplex *U, MyComplex *X, MyComplex *Y, MyComplex *TRF, MyComplex *R)
{
  #ifdef SIMPLE
    CUSOLVERERR(hipsolverDnCpotrf(handle, HIPBLAS_FILL_MODE_UPPER, NChan, U, NChan, TRF,   SizeTRF,  Info));
    CUSOLVERERR(hipsolverDnCpotrs(handle, HIPBLAS_FILL_MODE_UPPER, NChan, NChan, U, NChan, X, NChan, Info));
    CUSOLVERERR(hipsolverDnCpotrs(handle, HIPBLAS_FILL_MODE_UPPER, NChan, NChan, U, NChan, Y, NChan, Info));
  #else
    CUSOLVERERR(hipsolverDnZpotrf(handle, HIPBLAS_FILL_MODE_UPPER, NChan, U, NChan, TRF,   SizeTRF,  Info));
    CUSOLVERERR(hipsolverDnZpotrs(handle, HIPBLAS_FILL_MODE_UPPER, NChan, NChan, U, NChan, X, NChan, Info));
    CUSOLVERERR(hipsolverDnZpotrs(handle, HIPBLAS_FILL_MODE_UPPER, NChan, NChan, U, NChan, Y, NChan, Info));
  #endif

  cTraceHalfCoale<<<NChan,  32>>>(NChan, X, Y); 
  cFinalTraceForKernel<<<1, 32>>>(NChan, X, Y, R);

  /* With CUBLAS 
  MyComplex Num, alfa, beta;
  alfa.x=1.0; alfa.y=0.0;
  beta.x=0.0; beta.y=0.0;

  // Compute Num=Trace[(U^-1 * X) * (U^-1 * Y)]. (U^-1 * X) es X y (U^-1 * Y) es Y.
  #ifdef SIMPLE
    // U = X^t
    CUBLASERR(hipblasCgeam(handleC, HIPBLAS_OP_T, HIPBLAS_OP_N, NChannels, NChannels, &alfa, X, NChannels, &beta, NULL, NChannels, U, NChannels));

    // Num=X^t*Y
    CUBLASERR(hipblasCdotu(handleC, NChannels*NChannels, U, 1, Y, 1, &Num));
  #else
    // U = X^t
    CUBLASERR(hipblasZgeam(handleC, HIPBLAS_OP_T, HIPBLAS_OP_N, NChannels, NChannels, &alfa, X, NChannels, &beta, NULL, NChannels, U, NChannels));

    // Num=X^t*Y
    CUBLASERR(hipblasZdotu(handleC, NChannels*NChannels, U, 1, Y, 1, &Num));
  #endif
  
  // Compute Y=(Num / Trace[(U^-1 * Y)]). (U^-1 * Y). The result is in Y[0]
  cFinalTraceForCublas<<<1, 32>>>(NChannels, Num, Y);
  */

  return 0;
}


int RealOneClassicStep(hipsolverHandle_t handle, const int NChan, const int SizeTRF, int *Info, MyType *U, MyType *X, MyType *Y, MyType *TRF, MyType *R)
{
  #ifdef SIMPLE
    CUSOLVERERR(hipsolverDnSpotrf(handle, HIPBLAS_FILL_MODE_UPPER, NChan, U, NChan, TRF,   SizeTRF,  Info));
    CUSOLVERERR(hipsolverDnSpotrs(handle, HIPBLAS_FILL_MODE_UPPER, NChan, NChan, U, NChan, X, NChan, Info));
    CUSOLVERERR(hipsolverDnSpotrs(handle, HIPBLAS_FILL_MODE_UPPER, NChan, NChan, U, NChan, Y, NChan, Info));
  #else
    CUSOLVERERR(hipsolverDnDpotrf(handle, HIPBLAS_FILL_MODE_UPPER, NChan, U, NChan, TRF,   SizeTRF,  Info));
    CUSOLVERERR(hipsolverDnDpotrs(handle, HIPBLAS_FILL_MODE_UPPER, NChan, NChan, U, NChan, X, NChan, Info));
    CUSOLVERERR(hipsolverDnDpotrs(handle, HIPBLAS_FILL_MODE_UPPER, NChan, NChan, U, NChan, Y, NChan, Info));
  #endif

  dTraceHalfCoale<<<NChan,  32>>>(NChan, X, Y); 
  dFinalTraceForKernel<<<1, 32>>>(NChan, X, Y, R);
  
  return 0;
}


int ComplexOneStreamStep(hipsolverHandle_t handle, hipStream_t stream, const int NChan, const int SizeTRF, int *Info, MyComplex *U, MyComplex *X, MyComplex *Y, MyComplex *TRF, MyComplex *R)
{
  #ifdef SIMPLE
    CUSOLVERERR(hipsolverDnCpotrf(handle, HIPBLAS_FILL_MODE_UPPER, NChan, U, NChan, TRF,   SizeTRF,  Info));
    CUSOLVERERR(hipsolverDnCpotrs(handle, HIPBLAS_FILL_MODE_UPPER, NChan, NChan, U, NChan, X, NChan, Info));
    CUSOLVERERR(hipsolverDnCpotrs(handle, HIPBLAS_FILL_MODE_UPPER, NChan, NChan, U, NChan, Y, NChan, Info));
  #else
    CUSOLVERERR(hipsolverDnZpotrf(handle, HIPBLAS_FILL_MODE_UPPER, NChan, U, NChan, TRF,   SizeTRF,  Info));
    CUSOLVERERR(hipsolverDnZpotrs(handle, HIPBLAS_FILL_MODE_UPPER, NChan, NChan, U, NChan, X, NChan, Info));
    CUSOLVERERR(hipsolverDnZpotrs(handle, HIPBLAS_FILL_MODE_UPPER, NChan, NChan, U, NChan, Y, NChan, Info));
  #endif

  cTraceHalfCoale<<<NChan,  32, 0, stream>>>(NChan, X, Y); 
  cFinalTraceForKernel<<<1, 32, 0, stream>>>(NChan, X, Y, R);
  
  return 0;
}


int RealOneStreamStep(hipsolverHandle_t handle, hipStream_t stream, const int NChan, const int SizeTRF, int *Info, MyType *U, MyType *X, MyType *Y, MyType *TRF, MyType *R)
{
  #ifdef SIMPLE
    CUSOLVERERR(hipsolverDnSpotrf(handle, HIPBLAS_FILL_MODE_UPPER, NChan, U, NChan, TRF,   SizeTRF,  Info));
    CUSOLVERERR(hipsolverDnSpotrs(handle, HIPBLAS_FILL_MODE_UPPER, NChan, NChan, U, NChan, X, NChan, Info));
    CUSOLVERERR(hipsolverDnSpotrs(handle, HIPBLAS_FILL_MODE_UPPER, NChan, NChan, U, NChan, Y, NChan, Info));
  #else
    CUSOLVERERR(hipsolverDnDpotrf(handle, HIPBLAS_FILL_MODE_UPPER, NChan, U, NChan, TRF,   SizeTRF,  Info));
    CUSOLVERERR(hipsolverDnDpotrs(handle, HIPBLAS_FILL_MODE_UPPER, NChan, NChan, U, NChan, X, NChan, Info));
    CUSOLVERERR(hipsolverDnDpotrs(handle, HIPBLAS_FILL_MODE_UPPER, NChan, NChan, U, NChan, Y, NChan, Info));
  #endif

  dTraceHalfCoale<<<NChan,  32, 0, stream>>>(NChan, X, Y); 
  dFinalTraceForKernel<<<1, 32, 0, stream>>>(NChan, X, Y, R);
  
  return 0;
}


int ComplexClassic(const int NChannels, const int NInverses, const MyComplex *A, const MyComplex *B, const MyComplex *C, MyComplex *Result)
{
  size_t     Size;
  int       *Err=NULL, SizeTRF, i;
  MyComplex *U=NULL, *X=NULL, *Y=NULL, *R=NULL, *TRF=NULL;

  hipsolverHandle_t handle=NULL;

  Size=(size_t)NChannels * (size_t)NChannels;

  CUSOLVERERR(hipsolverDnCreate(&handle));

  CUDAERR(hipMalloc((void **)&U, sizeof(MyComplex)*Size));
  CUDAERR(hipMalloc((void **)&X, sizeof(MyComplex)*Size));
  CUDAERR(hipMalloc((void **)&Y, sizeof(MyComplex)*Size));
  CUDAERR(hipMalloc((void **)&R, sizeof(MyComplex)*NInverses));

  CUDAERR(hipMemcpy(U, A, sizeof(MyComplex)*Size, hipMemcpyHostToDevice));
  CUDAERR(hipMemcpy(X, B, sizeof(MyComplex)*Size, hipMemcpyHostToDevice));
  CUDAERR(hipMemcpy(Y, C, sizeof(MyComplex)*Size, hipMemcpyHostToDevice));

  #ifdef SIMPLE
    CUSOLVERERR(hipsolverDnCpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, NChannels, U, NChannels, &SizeTRF));
  #else
    CUSOLVERERR(hipsolverDnZpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, NChannels, U, NChannels, &SizeTRF));
  #endif
  CUDAERR(hipMalloc((void **)&TRF, sizeof(MyComplex)*SizeTRF));
  CUDAERR(hipMalloc((void **)&Err, sizeof(int)));

  ComplexOneClassicStep(handle, NChannels, SizeTRF, Err, U, X, Y, TRF, &R[0]);

  for(i=1; i<NInverses; i++)
  {
    CUDAERR(hipMemcpy(U, &A[Size*i], sizeof(MyComplex)*Size, hipMemcpyHostToDevice));
    CUDAERR(hipMemcpy(X, &B[Size*i], sizeof(MyComplex)*Size, hipMemcpyHostToDevice));
    CUDAERR(hipMemcpy(Y, &C[Size*i], sizeof(MyComplex)*Size, hipMemcpyHostToDevice));

    ComplexOneClassicStep(handle, NChannels, SizeTRF, Err, U, X, Y, TRF, &R[i]);
  }
  CUDAERR(hipMemcpy(Result, R, sizeof(MyComplex)*NInverses, hipMemcpyDeviceToHost));

  CUDAERR(hipFree(TRF)); CUDAERR(hipFree(Err)); CUDAERR(hipFree(Y));
  CUDAERR(hipFree(U));   CUDAERR(hipFree(X));   CUDAERR(hipFree(R));

  CUSOLVERERR(hipsolverDnDestroy(handle));
  
  return 0;
}


int RealClassic(const int NChannels, const int NInverses, const MyType *A, const MyType *B, const MyType *C, MyType *Result)
{
  size_t  Size;
  int     *Err=NULL, SizeTRF, i;
  MyType  *U=NULL, *X=NULL, *Y=NULL, *R=NULL, *TRF=NULL;

  hipsolverHandle_t handle=NULL;

  Size=(size_t)NChannels * (size_t)NChannels;

  CUSOLVERERR(hipsolverDnCreate(&handle));

  CUDAERR(hipMalloc((void **)&U, sizeof(MyType)*Size));
  CUDAERR(hipMalloc((void **)&X, sizeof(MyType)*Size));
  CUDAERR(hipMalloc((void **)&Y, sizeof(MyType)*Size));
  CUDAERR(hipMalloc((void **)&R, sizeof(MyType)*NInverses));

  CUDAERR(hipMemcpy(U, A, sizeof(MyType)*Size, hipMemcpyHostToDevice));
  CUDAERR(hipMemcpy(X, B, sizeof(MyType)*Size, hipMemcpyHostToDevice));
  CUDAERR(hipMemcpy(Y, C, sizeof(MyType)*Size, hipMemcpyHostToDevice));

  #ifdef SIMPLE
    CUSOLVERERR(hipsolverDnSpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, NChannels, U, NChannels, &SizeTRF));
  #else
    CUSOLVERERR(hipsolverDnDpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, NChannels, U, NChannels, &SizeTRF));
  #endif
  CUDAERR(hipMalloc((void **)&TRF, sizeof(MyType)*SizeTRF));
  CUDAERR(hipMalloc((void **)&Err, sizeof(int)));

  RealOneClassicStep(handle, NChannels, SizeTRF, Err, U, X, Y, TRF, &R[0]);

  for(i=1; i<NInverses; i++)
  {
    CUDAERR(hipMemcpy(U, &A[Size*i], sizeof(MyType)*Size, hipMemcpyHostToDevice));
    CUDAERR(hipMemcpy(X, &B[Size*i], sizeof(MyType)*Size, hipMemcpyHostToDevice));
    CUDAERR(hipMemcpy(Y, &C[Size*i], sizeof(MyType)*Size, hipMemcpyHostToDevice));

    RealOneClassicStep(handle, NChannels, SizeTRF, Err, U, X, Y, TRF, &R[i]);
  }
  CUDAERR(hipMemcpy(Result, R, sizeof(MyType)*NInverses, hipMemcpyDeviceToHost));

  CUDAERR(hipFree(TRF)); CUDAERR(hipFree(Err)); CUDAERR(hipFree(Y));
  CUDAERR(hipFree(U));   CUDAERR(hipFree(X));   CUDAERR(hipFree(R));

  CUSOLVERERR(hipsolverDnDestroy(handle));
  
  return 0;
}


int ComplexZeroCopy(const int NChannels, const int NInverses, MyComplex *A, MyComplex *B, MyComplex *C, MyComplex *Result)
{
  size_t    Size;
  int       *Err=NULL, SizeTRF, i;
  MyComplex *U=NULL, *X=NULL, *Y=NULL, *R=NULL, *TRF=NULL;

  hipsolverHandle_t handle=NULL;

  Size=(size_t)NChannels * (size_t)NChannels;

  CUDAERR(hipSetDeviceFlags(hipDeviceMapHost));

  CUSOLVERERR(hipsolverDnCreate(&handle));

  CUDAERR(hipHostGetDevicePointer((void **)&U, (void *)A, 0));
  CUDAERR(hipHostGetDevicePointer((void **)&X, (void *)B, 0));
  CUDAERR(hipHostGetDevicePointer((void **)&Y, (void *)C, 0));

  CUDAERR(hipMalloc((void **)&R, sizeof(MyComplex)*NInverses));

  #ifdef SIMPLE
    CUSOLVERERR(hipsolverDnCpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, NChannels, U, NChannels, &SizeTRF));
  #else
    CUSOLVERERR(hipsolverDnZpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, NChannels, U, NChannels, &SizeTRF));
  #endif
  CUDAERR(hipMalloc((void **)&TRF, sizeof(MyComplex)*SizeTRF));
  CUDAERR(hipMalloc((void **)&Err, sizeof(int)));

  ComplexOneClassicStep(handle, NChannels, SizeTRF, Err, U, X, Y, TRF, &R[0]);

  for(i=1; i<NInverses; i++)
    ComplexOneClassicStep(handle, NChannels, SizeTRF, Err, &U[Size*i], &X[Size*i], &Y[Size*i], TRF, &R[i]);

  CUDAERR(hipMemcpy(Result, R, sizeof(MyComplex)*NInverses, hipMemcpyDeviceToHost));

  CUDAERR(hipFree(TRF)); CUDAERR(hipFree(Err)); CUDAERR(hipFree(R));

  CUSOLVERERR(hipsolverDnDestroy(handle));
  
  return 0;
}


int RealZeroCopy(const int NChannels, const int NInverses, MyType *A, MyType *B, MyType *C, MyType *Result)
{
  size_t  Size;
  int     *Err=NULL, SizeTRF, i;
  MyType  *U=NULL, *X=NULL, *Y=NULL, *R=NULL, *TRF=NULL;

  hipsolverHandle_t handle=NULL;

  Size=(size_t)NChannels * (size_t)NChannels;

  CUDAERR(hipSetDeviceFlags(hipDeviceMapHost));

  CUSOLVERERR(hipsolverDnCreate(&handle));

  CUDAERR(hipHostGetDevicePointer((void **)&U, (void *)A, 0));
  CUDAERR(hipHostGetDevicePointer((void **)&X, (void *)B, 0));
  CUDAERR(hipHostGetDevicePointer((void **)&Y, (void *)C, 0));

  CUDAERR(hipMalloc((void **)&R, sizeof(MyType)*NInverses));

  #ifdef SIMPLE
    CUSOLVERERR(hipsolverDnSpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, NChannels, U, NChannels, &SizeTRF));
  #else
    CUSOLVERERR(hipsolverDnDpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, NChannels, U, NChannels, &SizeTRF));
  #endif
  CUDAERR(hipMalloc((void **)&TRF, sizeof(MyType)*SizeTRF));
  CUDAERR(hipMalloc((void **)&Err, sizeof(int)));

  RealOneClassicStep(handle, NChannels, SizeTRF, Err, U, X, Y, TRF, &R[0]);

  for(i=1; i<NInverses; i++)
    RealOneClassicStep(handle, NChannels, SizeTRF, Err, &U[Size*i], &X[Size*i], &Y[Size*i], TRF, &R[i]);

  CUDAERR(hipMemcpy(Result, R, sizeof(MyType)*NInverses, hipMemcpyDeviceToHost));

  CUDAERR(hipFree(TRF)); CUDAERR(hipFree(Err)); CUDAERR(hipFree(R));

  CUSOLVERERR(hipsolverDnDestroy(handle));
  
  return 0;
}


int ComplexOpenMP(const int NChannels, const int NInverses, const MyComplex *A, const MyComplex *B, const MyComplex *C, MyComplex *Result, const int Cores)
{
  MyComplex *R=NULL;

  CUDAERR(hipMalloc((void **)&R, sizeof(MyComplex)*NInverses));

  #ifdef OMP
    #pragma omp parallel num_threads(Cores)
  {
  #endif
    size_t    Size;
    int       *Err=NULL, SizeTRF, i, Chunk, MyID=0, NumTh=1, Start, End;
    MyComplex *U=NULL, *X=NULL, *Y=NULL, *TRF=NULL;

    hipsolverHandle_t handle;
    hipStream_t       stream;

    Size=(size_t)NChannels * (size_t)NChannels;

    CUSOLVERERROMP(hipsolverDnCreate(&handle));
    CUDAERROMP(hipStreamCreate(&stream));
    CUSOLVERERROMP(hipsolverSetStream(handle, stream));
    
    #ifdef OMP
      MyID  = omp_get_thread_num();
      NumTh = omp_get_num_threads();
    #endif
    Chunk = NInverses / NumTh;
    Start = MyID * Chunk;

    if (MyID==(NumTh-1)) { End=Start + Chunk + (NInverses % NumTh); } else { End=Start + Chunk; }

    CUDAERROMP(hipMalloc((void **)&U, sizeof(MyComplex)*Size));
    CUDAERROMP(hipMalloc((void **)&X, sizeof(MyComplex)*Size));
    CUDAERROMP(hipMalloc((void **)&Y, sizeof(MyComplex)*Size));

    CUDAERROMP(hipMemcpyAsync(U, &A[Start*Size], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, stream));
    CUDAERROMP(hipMemcpyAsync(X, &B[Start*Size], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, stream));
    CUDAERROMP(hipMemcpyAsync(Y, &C[Start*Size], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, stream));

    #ifdef SIMPLE
      CUSOLVERERROMP(hipsolverDnCpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, NChannels, U, NChannels, &SizeTRF));
    #else
      CUSOLVERERROMP(hipsolverDnZpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, NChannels, U, NChannels, &SizeTRF));
    #endif
    CUDAERROMP(hipMalloc((void **)&TRF, sizeof(MyComplex)*SizeTRF));
    CUDAERROMP(hipMalloc((void **)&Err, sizeof(int)));

    ComplexOneStreamStep(handle, stream, NChannels, SizeTRF, Err, U, X, Y, TRF, &R[Start]);

    for(i=(Start+1); i<End; i++)
    {
      CUDAERROMP(hipMemcpyAsync(U, &A[Size*i], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, stream));
      CUDAERROMP(hipMemcpyAsync(X, &B[Size*i], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, stream));
      CUDAERROMP(hipMemcpyAsync(Y, &C[Size*i], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, stream));

      ComplexOneStreamStep(handle, stream, NChannels, SizeTRF, Err, U, X, Y, TRF, &R[i]);
    }
    CUDAERROMP(hipStreamSynchronize(stream));
    CUSOLVERERROMP(hipsolverDnDestroy(handle));
    CUDAERROMP(hipStreamDestroy(stream));

    CUDAERROMP(hipFree(TRF)); CUDAERROMP(hipFree(Err)); CUDAERROMP(hipFree(Y));
    CUDAERROMP(hipFree(U));   CUDAERROMP(hipFree(X));
  #ifdef OMP
  }
  #endif  
  CUDAERROMP(hipMemcpy(Result, R, sizeof(MyComplex)*NInverses, hipMemcpyDeviceToHost));
  CUDAERROMP(hipFree(R)); 

  return 0;
}


int RealOpenMP(const int NChannels, const int NInverses, const MyType *A, const MyType *B, const MyType *C, MyType *Result, const int Cores)
{
  MyType *R=NULL;

  CUDAERR(hipMalloc((void **)&R, sizeof(MyType)*NInverses));

  #ifdef OMP
    #pragma omp parallel num_threads(Cores)
  {
  #endif
    size_t  Size;
    int     *Err=NULL, SizeTRF, i, Chunk, MyID=0, NumTh=1, Start, End;
    MyType  *U=NULL, *X=NULL, *Y=NULL, *TRF=NULL;

    hipsolverHandle_t handle;
    hipStream_t       stream;

    Size=(size_t)NChannels * (size_t)NChannels;

    CUSOLVERERROMP(hipsolverDnCreate(&handle));
    CUDAERROMP(hipStreamCreate(&stream));
    CUSOLVERERROMP(hipsolverSetStream(handle, stream));
    
    #ifdef OMP
      MyID  = omp_get_thread_num();
      NumTh = omp_get_num_threads();
    #endif
    Chunk = NInverses / NumTh;
    Start = MyID * Chunk;

    if (MyID==(NumTh-1)) { End=Start + Chunk + (NInverses % NumTh); } else { End=Start + Chunk; }

    CUDAERROMP(hipMalloc((void **)&U, sizeof(MyType)*Size));
    CUDAERROMP(hipMalloc((void **)&X, sizeof(MyType)*Size));
    CUDAERROMP(hipMalloc((void **)&Y, sizeof(MyType)*Size));

    CUDAERROMP(hipMemcpyAsync(U, &A[Start*Size], sizeof(MyType)*Size, hipMemcpyHostToDevice, stream));
    CUDAERROMP(hipMemcpyAsync(X, &B[Start*Size], sizeof(MyType)*Size, hipMemcpyHostToDevice, stream));
    CUDAERROMP(hipMemcpyAsync(Y, &C[Start*Size], sizeof(MyType)*Size, hipMemcpyHostToDevice, stream));

    #ifdef SIMPLE
      CUSOLVERERROMP(hipsolverDnSpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, NChannels, U, NChannels, &SizeTRF));
    #else
      CUSOLVERERROMP(hipsolverDnDpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, NChannels, U, NChannels, &SizeTRF));
    #endif
    CUDAERROMP(hipMalloc((void **)&TRF, sizeof(MyType)*SizeTRF));
    CUDAERROMP(hipMalloc((void **)&Err, sizeof(int)));

    RealOneStreamStep(handle, stream, NChannels, SizeTRF, Err, U, X, Y, TRF, &R[Start]);

    for(i=(Start+1); i<End; i++)
    {
      CUDAERROMP(hipMemcpyAsync(U, &A[Size*i], sizeof(MyType)*Size, hipMemcpyHostToDevice, stream));
      CUDAERROMP(hipMemcpyAsync(X, &B[Size*i], sizeof(MyType)*Size, hipMemcpyHostToDevice, stream));
      CUDAERROMP(hipMemcpyAsync(Y, &C[Size*i], sizeof(MyType)*Size, hipMemcpyHostToDevice, stream));

      RealOneStreamStep(handle, stream, NChannels, SizeTRF, Err, U, X, Y, TRF, &R[i]);
    }
    CUDAERROMP(hipStreamSynchronize(stream));
    CUSOLVERERROMP(hipsolverDnDestroy(handle));
    CUDAERROMP(hipStreamDestroy(stream));

    CUDAERROMP(hipFree(TRF)); CUDAERROMP(hipFree(Err)); CUDAERROMP(hipFree(Y));
    CUDAERROMP(hipFree(U));   CUDAERROMP(hipFree(X));
  #ifdef OMP
  }
  #endif  
  CUDAERROMP(hipMemcpy(Result, R, sizeof(MyType)*NInverses, hipMemcpyDeviceToHost));
  CUDAERROMP(hipFree(R)); 

  return 0;
}


int ComplexStreams(const int NChannels, const int NInverses, const MyComplex *A, const MyComplex *B, const MyComplex *C, MyComplex *Result, const int NStreams)
{
  size_t    Size;
  int       *Err=NULL, SizeTRF, i, j;
  MyComplex *U=NULL, *X=NULL, *Y=NULL, *R=NULL, *TRF=NULL;

  hipsolverHandle_t handles[NStreams];
  hipStream_t       streams[NStreams];

  for (i=0; i<NStreams; i++)
  {
    CUDAERR(hipStreamCreate(&streams[i]));
    CUSOLVERERR(hipsolverDnCreate(&handles[i]));
    CUSOLVERERR(hipsolverSetStream(handles[i], streams[i]));
  }  
  Size=(size_t)NChannels * (size_t)NChannels;

  CUDAERR(hipMalloc((void **)&U, sizeof(MyComplex)*Size*NStreams));
  CUDAERR(hipMalloc((void **)&X, sizeof(MyComplex)*Size*NStreams));
  CUDAERR(hipMalloc((void **)&Y, sizeof(MyComplex)*Size*NStreams));
  CUDAERR(hipMalloc((void **)&R, sizeof(MyComplex)*NInverses));

  CUDAERR(hipMemcpy(U, A, sizeof(MyComplex)*Size, hipMemcpyHostToDevice));
  CUDAERR(hipMemcpy(X, B, sizeof(MyComplex)*Size, hipMemcpyHostToDevice));
  CUDAERR(hipMemcpy(Y, C, sizeof(MyComplex)*Size, hipMemcpyHostToDevice));
  #ifdef SIMPLE
    CUSOLVERERR(hipsolverDnCpotrf_bufferSize(handles[0], HIPBLAS_FILL_MODE_UPPER, NChannels, U, NChannels, &SizeTRF));
  #else
    CUSOLVERERR(hipsolverDnZpotrf_bufferSize(handles[0], HIPBLAS_FILL_MODE_UPPER, NChannels, U, NChannels, &SizeTRF));
  #endif
  CUDAERR(hipMalloc((void **)&TRF, sizeof(MyComplex)*SizeTRF*NStreams));
  CUDAERR(hipMalloc((void **)&Err, sizeof(int)*NStreams));

  for(i=0; i<(NInverses-NStreams); i+=NStreams)
  {
    for(j=0; j<NStreams; j++)
    {
      CUDAERR(hipMemcpyAsync(&U[Size*j], &A[Size*(i+j)], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, streams[j]));
      CUDAERR(hipMemcpyAsync(&X[Size*j], &B[Size*(i+j)], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, streams[j]));
      CUDAERR(hipMemcpyAsync(&Y[Size*j], &C[Size*(i+j)], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, streams[j]));

      ComplexOneStreamStep(handles[j], streams[j], NChannels, SizeTRF, &Err[j], &U[Size*j], &X[Size*j], &Y[Size*j], &TRF[SizeTRF*j], &R[i+j]);
    }
  }
  for (j=0; j<(NInverses-i); j++)
  {
    CUDAERR(hipMemcpyAsync(&U[Size*j], &A[Size*(i+j)], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, streams[j]));
    CUDAERR(hipMemcpyAsync(&X[Size*j], &B[Size*(i+j)], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, streams[j]));
    CUDAERR(hipMemcpyAsync(&Y[Size*j], &C[Size*(i+j)], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, streams[j]));

    ComplexOneStreamStep(handles[j], streams[j], NChannels, SizeTRF, &Err[j], &U[Size*j], &X[Size*j], &Y[Size*j], &TRF[SizeTRF*j], &R[i+j]);
  }
  for (i=0; i<NStreams; i++)
    CUDAERR(hipStreamSynchronize(streams[i]));
  CUDAERR(hipMemcpy(Result, R, sizeof(MyComplex)*NInverses, hipMemcpyDeviceToHost));
  for (i=0; i<NStreams; i++)
  {
    CUSOLVERERR(hipsolverDnDestroy(handles[i]));
    CUDAERR(hipStreamDestroy(streams[i]));
  }

  CUDAERR(hipFree(TRF)); CUDAERR(hipFree(Err)); CUDAERR(hipFree(Y));
  CUDAERR(hipFree(U));   CUDAERR(hipFree(X));   CUDAERR(hipFree(R));

  return 0;
}


int RealStreams(const int NChannels, const int NInverses, const MyType *A, const MyType *B, const MyType *C, MyType *Result, const int NStreams)
{
  size_t  Size;
  int     *Err=NULL, SizeTRF, i, j;
  MyType  *U=NULL, *X=NULL, *Y=NULL, *R=NULL, *TRF=NULL;

  hipsolverHandle_t handles[NStreams];
  hipStream_t       streams[NStreams];

  for (i=0; i<NStreams; i++)
  {
    CUDAERR(hipStreamCreate(&streams[i]));
    CUSOLVERERR(hipsolverDnCreate(&handles[i]));
    CUSOLVERERR(hipsolverSetStream(handles[i], streams[i]));
  }  
  Size=(size_t)NChannels * (size_t)NChannels;

  CUDAERR(hipMalloc((void **)&U, sizeof(MyType)*Size*NStreams));
  CUDAERR(hipMalloc((void **)&X, sizeof(MyType)*Size*NStreams));
  CUDAERR(hipMalloc((void **)&Y, sizeof(MyType)*Size*NStreams));
  CUDAERR(hipMalloc((void **)&R, sizeof(MyType)*NInverses));

  CUDAERR(hipMemcpy(U, A, sizeof(MyType)*Size, hipMemcpyHostToDevice));
  CUDAERR(hipMemcpy(X, B, sizeof(MyType)*Size, hipMemcpyHostToDevice));
  CUDAERR(hipMemcpy(Y, C, sizeof(MyType)*Size, hipMemcpyHostToDevice));
  #ifdef SIMPLE
    CUSOLVERERR(hipsolverDnSpotrf_bufferSize(handles[0], HIPBLAS_FILL_MODE_UPPER, NChannels, U, NChannels, &SizeTRF));
  #else
    CUSOLVERERR(hipsolverDnDpotrf_bufferSize(handles[0], HIPBLAS_FILL_MODE_UPPER, NChannels, U, NChannels, &SizeTRF));
  #endif
  CUDAERR(hipMalloc((void **)&TRF, sizeof(MyType)*SizeTRF*NStreams));
  CUDAERR(hipMalloc((void **)&Err, sizeof(int)*NStreams));

  for(i=0; i<(NInverses-NStreams); i+=NStreams)
  {
    for(j=0; j<NStreams; j++)
    {
      CUDAERR(hipMemcpyAsync(&U[Size*j], &A[Size*(i+j)], sizeof(MyType)*Size, hipMemcpyHostToDevice, streams[j]));
      CUDAERR(hipMemcpyAsync(&X[Size*j], &B[Size*(i+j)], sizeof(MyType)*Size, hipMemcpyHostToDevice, streams[j]));
      CUDAERR(hipMemcpyAsync(&Y[Size*j], &C[Size*(i+j)], sizeof(MyType)*Size, hipMemcpyHostToDevice, streams[j]));

      RealOneStreamStep(handles[j], streams[j], NChannels, SizeTRF, &Err[j], &U[Size*j], &X[Size*j], &Y[Size*j], &TRF[SizeTRF*j], &R[i+j]);
    }
  }
  for (j=0; j<(NInverses-i); j++)
  {
    CUDAERR(hipMemcpyAsync(&U[Size*j], &A[Size*(i+j)], sizeof(MyType)*Size, hipMemcpyHostToDevice, streams[j]));
    CUDAERR(hipMemcpyAsync(&X[Size*j], &B[Size*(i+j)], sizeof(MyType)*Size, hipMemcpyHostToDevice, streams[j]));
    CUDAERR(hipMemcpyAsync(&Y[Size*j], &C[Size*(i+j)], sizeof(MyType)*Size, hipMemcpyHostToDevice, streams[j]));

    RealOneStreamStep(handles[j], streams[j], NChannels, SizeTRF, &Err[j], &U[Size*j], &X[Size*j], &Y[Size*j], &TRF[SizeTRF*j], &R[i+j]);
  }
  for (i=0; i<NStreams; i++)
    CUDAERR(hipStreamSynchronize(streams[i]));
  CUDAERR(hipMemcpy(Result, R, sizeof(MyType)*NInverses, hipMemcpyDeviceToHost));
  for (i=0; i<NStreams; i++)
  {
    CUSOLVERERR(hipsolverDnDestroy(handles[i]));
    CUDAERR(hipStreamDestroy(streams[i]));
  }

  CUDAERR(hipFree(TRF)); CUDAERR(hipFree(Err)); CUDAERR(hipFree(Y));
  CUDAERR(hipFree(U));   CUDAERR(hipFree(X));   CUDAERR(hipFree(R));

  return 0;
}


int ComplexEvents(const int NChannels, const int NInverses, const MyComplex *A, const MyComplex *B, const MyComplex *C, MyComplex *Result)
{
  size_t     Size;
  int        *Err=NULL, SizeTRF, i;
  MyComplex  *U1=NULL, *X1=NULL, *Y1=NULL, *U2=NULL, *X2=NULL, *Y2=NULL, *R=NULL, *TRF=NULL;

  hipsolverHandle_t handle;
  hipStream_t       stream1, stream2;
  hipEvent_t        evento1, evento2;

  CUDAERR(hipStreamCreate(&stream1));
  CUDAERR(hipStreamCreate(&stream2));
  CUDAERR(hipEventCreate(&evento1));
  CUDAERR(hipEventCreate(&evento2));
  CUSOLVERERR(hipsolverDnCreate(&handle));
  CUSOLVERERR(hipsolverSetStream(handle, stream2));

  Size=(size_t)NChannels * (size_t)NChannels;

  CUDAERR(hipMalloc((void **)&U1, sizeof(MyComplex)*Size));
  CUDAERR(hipMalloc((void **)&X1, sizeof(MyComplex)*Size));
  CUDAERR(hipMalloc((void **)&Y1, sizeof(MyComplex)*Size));
  CUDAERR(hipMalloc((void **)&U2, sizeof(MyComplex)*Size));
  CUDAERR(hipMalloc((void **)&X2, sizeof(MyComplex)*Size));
  CUDAERR(hipMalloc((void **)&Y2, sizeof(MyComplex)*Size));
  CUDAERR(hipMalloc((void **)&R,  sizeof(MyComplex)*NInverses));

  CUDAERR(hipMemcpy(U1, A, sizeof(MyComplex)*Size, hipMemcpyHostToDevice));
  CUDAERR(hipMemcpy(X1, B, sizeof(MyComplex)*Size, hipMemcpyHostToDevice));
  CUDAERR(hipMemcpy(Y1, C, sizeof(MyComplex)*Size, hipMemcpyHostToDevice));
  #ifdef SIMPLE
    CUSOLVERERR(hipsolverDnCpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, NChannels, U1, NChannels, &SizeTRF));
  #else
    CUSOLVERERR(hipsolverDnZpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, NChannels, U1, NChannels, &SizeTRF));
  #endif
  CUDAERR(hipMalloc((void **)&TRF, sizeof(MyComplex)*SizeTRF));
  CUDAERR(hipMalloc((void **)&Err, sizeof(int)));

  hipEventRecord(evento1, stream1);
  for (i=1; i<NInverses; i++)
  {
    if ((i % 2) != 0)
    {
      hipStreamWaitEvent(stream2, evento1);
      ComplexOneStreamStep(handle, stream2, NChannels, SizeTRF, Err, U1, X1, Y1, TRF, &R[i-1]);
      hipEventRecord(evento2, stream2);

      CUDAERR(hipMemcpyAsync(U2, &A[Size*i], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, stream1));
      CUDAERR(hipMemcpyAsync(X2, &B[Size*i], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, stream1));
      CUDAERR(hipMemcpyAsync(Y2, &C[Size*i], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, stream1));
      hipEventRecord(evento1, stream1);
      hipStreamWaitEvent(stream1, evento2);
    } else {
      hipStreamWaitEvent(stream2, evento1);
      ComplexOneStreamStep(handle, stream2, NChannels, SizeTRF, Err, U2, X2, Y2, TRF, &R[i-1]);
      hipEventRecord(evento2, stream2);

      CUDAERR(hipMemcpyAsync(U1, &A[Size*i], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, stream1));
      CUDAERR(hipMemcpyAsync(X1, &B[Size*i], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, stream1));
      CUDAERR(hipMemcpyAsync(Y1, &C[Size*i], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, stream1));
      hipEventRecord(evento1, stream1);
      hipStreamWaitEvent(stream1, evento2);
    }
  }
  hipStreamWaitEvent(stream2, evento1);
  if ((NInverses % 2) == 0)
    ComplexOneStreamStep(handle, stream2, NChannels, SizeTRF, Err, U2, X2, Y2, TRF, &R[NInverses-1]);
  else
    ComplexOneStreamStep(handle, stream2, NChannels, SizeTRF, Err, U1, X1, Y1, TRF, &R[NInverses-1]);

  CUDAERR(hipStreamSynchronize(stream1));
  CUDAERR(hipStreamSynchronize(stream2));

  CUDAERR(hipMemcpy(Result, R, sizeof(MyComplex)*NInverses, hipMemcpyDeviceToHost));

  CUSOLVERERR(hipsolverDnDestroy(handle));
  CUDAERR(hipStreamDestroy(stream1));
  CUDAERR(hipStreamDestroy(stream2));
  CUDAERR(hipEventDestroy(evento1));
  CUDAERR(hipEventDestroy(evento2));

  CUDAERR(hipFree(TRF)); CUDAERR(hipFree(Err)); CUDAERR(hipFree(Y1)); CUDAERR(hipFree(U1));
  CUDAERR(hipFree(X1));  CUDAERR(hipFree(Y2));  CUDAERR(hipFree(U2)); CUDAERR(hipFree(X2));
  CUDAERR(hipFree(R));

  return 0;
}


int RealEvents(const int NChannels, const int NInverses, const MyType *A, const MyType *B, const MyType *C, MyType *Result)
{
  size_t  Size;
  int     *Err=NULL, SizeTRF, i;
  MyType  *U1=NULL, *X1=NULL, *Y1=NULL, *U2=NULL, *X2=NULL, *Y2=NULL, *R=NULL, *TRF=NULL;

  hipsolverHandle_t handle;
  hipStream_t       stream1, stream2;
  hipEvent_t        evento1, evento2;

  CUDAERR(hipStreamCreate(&stream1));
  CUDAERR(hipStreamCreate(&stream2));
  CUDAERR(hipEventCreate(&evento1));
  CUDAERR(hipEventCreate(&evento2));
  CUSOLVERERR(hipsolverDnCreate(&handle));
  CUSOLVERERR(hipsolverSetStream(handle, stream2));

  Size=(size_t)NChannels * (size_t)NChannels;

  CUDAERR(hipMalloc((void **)&U1, sizeof(MyType)*Size));
  CUDAERR(hipMalloc((void **)&X1, sizeof(MyType)*Size));
  CUDAERR(hipMalloc((void **)&Y1, sizeof(MyType)*Size));
  CUDAERR(hipMalloc((void **)&U2, sizeof(MyType)*Size));
  CUDAERR(hipMalloc((void **)&X2, sizeof(MyType)*Size));
  CUDAERR(hipMalloc((void **)&Y2, sizeof(MyType)*Size));
  CUDAERR(hipMalloc((void **)&R,  sizeof(MyType)*NInverses));

  CUDAERR(hipMemcpy(U1, A, sizeof(MyType)*Size, hipMemcpyHostToDevice));
  CUDAERR(hipMemcpy(X1, B, sizeof(MyType)*Size, hipMemcpyHostToDevice));
  CUDAERR(hipMemcpy(Y1, C, sizeof(MyType)*Size, hipMemcpyHostToDevice));
  #ifdef SIMPLE
    CUSOLVERERR(hipsolverDnSpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, NChannels, U1, NChannels, &SizeTRF));
  #else
    CUSOLVERERR(hipsolverDnDpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, NChannels, U1, NChannels, &SizeTRF));
  #endif
  CUDAERR(hipMalloc((void **)&TRF, sizeof(MyType)*SizeTRF));
  CUDAERR(hipMalloc((void **)&Err, sizeof(int)));

  hipEventRecord(evento1, stream1);
  for (i=1; i<NInverses; i++)
  {
    if ((i % 2) != 0)
    {
      hipStreamWaitEvent(stream2, evento1);
      RealOneStreamStep(handle, stream2, NChannels, SizeTRF, Err, U1, X1, Y1, TRF, &R[i-1]);
      hipEventRecord(evento2, stream2);

      CUDAERR(hipMemcpyAsync(U2, &A[Size*i], sizeof(MyType)*Size, hipMemcpyHostToDevice, stream1));
      CUDAERR(hipMemcpyAsync(X2, &B[Size*i], sizeof(MyType)*Size, hipMemcpyHostToDevice, stream1));
      CUDAERR(hipMemcpyAsync(Y2, &C[Size*i], sizeof(MyType)*Size, hipMemcpyHostToDevice, stream1));
      hipEventRecord(evento1, stream1);
      hipStreamWaitEvent(stream1, evento2);
    } else {
      hipStreamWaitEvent(stream2, evento1);
      RealOneStreamStep(handle, stream2, NChannels, SizeTRF, Err, U2, X2, Y2, TRF, &R[i-1]);
      hipEventRecord(evento2, stream2);

      CUDAERR(hipMemcpyAsync(U1, &A[Size*i], sizeof(MyType)*Size, hipMemcpyHostToDevice, stream1));
      CUDAERR(hipMemcpyAsync(X1, &B[Size*i], sizeof(MyType)*Size, hipMemcpyHostToDevice, stream1));
      CUDAERR(hipMemcpyAsync(Y1, &C[Size*i], sizeof(MyType)*Size, hipMemcpyHostToDevice, stream1));
      hipEventRecord(evento1, stream1);
      hipStreamWaitEvent(stream1, evento2);
    }
  }
  hipStreamWaitEvent(stream2, evento1);
  if ((NInverses % 2) == 0)
    RealOneStreamStep(handle, stream2, NChannels, SizeTRF, Err, U2, X2, Y2, TRF, &R[NInverses-1]);
  else
    RealOneStreamStep(handle, stream2, NChannels, SizeTRF, Err, U1, X1, Y1, TRF, &R[NInverses-1]);

  CUDAERR(hipStreamSynchronize(stream1));
  CUDAERR(hipStreamSynchronize(stream2));

  CUDAERR(hipMemcpy(Result, R, sizeof(MyType)*NInverses, hipMemcpyDeviceToHost));

  CUSOLVERERR(hipsolverDnDestroy(handle));
  CUDAERR(hipStreamDestroy(stream1));
  CUDAERR(hipStreamDestroy(stream2));
  CUDAERR(hipEventDestroy(evento1));
  CUDAERR(hipEventDestroy(evento2));

  CUDAERR(hipFree(TRF)); CUDAERR(hipFree(Err)); CUDAERR(hipFree(Y1)); CUDAERR(hipFree(U1));
  CUDAERR(hipFree(X1));  CUDAERR(hipFree(Y2));  CUDAERR(hipFree(U2)); CUDAERR(hipFree(X2));
  CUDAERR(hipFree(R));

  return 0;
}


int ComplexEventsOMP(const int NChannels, const int NInverses, const MyComplex *A, const MyComplex *B, const MyComplex *C, MyComplex *Result,  const int Cores)
{
  MyComplex *R=NULL;

  CUDAERR(hipMalloc((void **)&R, sizeof(MyComplex)*NInverses));

  #ifdef OMP
    #pragma omp parallel num_threads(Cores)
  {
  #endif
    size_t    Size;
    int       *Err=NULL, SizeTRF, i, Chunk, MyID=0, NumTh=1, Start, End;
    MyComplex *U1=NULL, *X1=NULL, *Y1=NULL, *U2=NULL, *X2=NULL, *Y2=NULL, *TRF=NULL;

    hipsolverHandle_t handle;
    hipStream_t       stream1, stream2;
    hipEvent_t        evento1, evento2;

    Size=(size_t)NChannels * (size_t)NChannels;

    CUDAERROMP(hipStreamCreate(&stream1));
    CUDAERROMP(hipStreamCreate(&stream2));
    CUDAERROMP(hipEventCreate(&evento1));
    CUDAERROMP(hipEventCreate(&evento2));
    CUSOLVERERROMP(hipsolverDnCreate(&handle));
    CUSOLVERERROMP(hipsolverSetStream(handle, stream2));

    #ifdef OMP
      MyID  = omp_get_thread_num();
      NumTh = omp_get_num_threads();
    #endif
    Chunk = NInverses / NumTh;
    Start = MyID * Chunk;

    if (MyID==(NumTh-1)) { End=Start + Chunk + (NInverses % NumTh); } else { End=Start + Chunk; }

    CUDAERROMP(hipMalloc((void **)&U1, sizeof(MyComplex)*Size));
    CUDAERROMP(hipMalloc((void **)&X1, sizeof(MyComplex)*Size));
    CUDAERROMP(hipMalloc((void **)&Y1, sizeof(MyComplex)*Size));
    CUDAERROMP(hipMalloc((void **)&U2, sizeof(MyComplex)*Size));
    CUDAERROMP(hipMalloc((void **)&X2, sizeof(MyComplex)*Size));
    CUDAERROMP(hipMalloc((void **)&Y2, sizeof(MyComplex)*Size));

    CUDAERROMP(hipMemcpy(U1, &A[Start*Size], sizeof(MyComplex)*Size, hipMemcpyHostToDevice));
    CUDAERROMP(hipMemcpy(X1, &B[Start*Size], sizeof(MyComplex)*Size, hipMemcpyHostToDevice));
    CUDAERROMP(hipMemcpy(Y1, &C[Start*Size], sizeof(MyComplex)*Size, hipMemcpyHostToDevice));

    #ifdef SIMPLE
      CUSOLVERERROMP(hipsolverDnCpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, NChannels, U1, NChannels, &SizeTRF));
    #else
      CUSOLVERERROMP(hipsolverDnZpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, NChannels, U1, NChannels, &SizeTRF));
    #endif
    CUDAERROMP(hipMalloc((void **)&TRF, sizeof(MyComplex)*SizeTRF));
    CUDAERROMP(hipMalloc((void **)&Err, sizeof(int)));

    hipEventRecord(evento1, stream1);
    if ((Start % 2) == 0)
    {
      for(i=(Start+1); i<End; i++)
      {
        if ((i % 2) != 0)
        {
          hipStreamWaitEvent(stream2, evento1);
          ComplexOneStreamStep(handle, stream2, NChannels, SizeTRF, Err, U1, X1, Y1, TRF, &R[i-1]);
          hipEventRecord(evento2, stream2);

          CUDAERROMP(hipMemcpyAsync(U2, &A[Size*i], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, stream1));
          CUDAERROMP(hipMemcpyAsync(X2, &B[Size*i], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, stream1));
          CUDAERROMP(hipMemcpyAsync(Y2, &C[Size*i], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, stream1));
          hipEventRecord(evento1, stream1);
          hipStreamWaitEvent(stream1, evento2);
        } else {
          hipStreamWaitEvent(stream2, evento1);
          ComplexOneStreamStep(handle, stream2, NChannels, SizeTRF, Err, U2, X2, Y2, TRF, &R[i-1]);
          hipEventRecord(evento2, stream2);

          CUDAERROMP(hipMemcpyAsync(U1, &A[Size*i], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, stream1));
          CUDAERROMP(hipMemcpyAsync(X1, &B[Size*i], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, stream1));
          CUDAERROMP(hipMemcpyAsync(Y1, &C[Size*i], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, stream1));
          hipEventRecord(evento1, stream1);
          hipStreamWaitEvent(stream1, evento2);
        }
      }
      hipStreamWaitEvent(stream2, evento1);
      if ((End % 2) == 0)
        ComplexOneStreamStep(handle, stream2, NChannels, SizeTRF, Err, U2, X2, Y2, TRF, &R[End-1]);
      else
        ComplexOneStreamStep(handle, stream2, NChannels, SizeTRF, Err, U1, X1, Y1, TRF, &R[End-1]);
    } else {
      for(i=(Start+1); i<End; i++)
      {
        if ((i % 2) == 0)
        {
          hipStreamWaitEvent(stream2, evento1);
          ComplexOneStreamStep(handle, stream2, NChannels, SizeTRF, Err, U1, X1, Y1, TRF, &R[i-1]);
          hipEventRecord(evento2, stream2);

          CUDAERROMP(hipMemcpyAsync(U2, &A[Size*i], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, stream1));
          CUDAERROMP(hipMemcpyAsync(X2, &B[Size*i], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, stream1));
          CUDAERROMP(hipMemcpyAsync(Y2, &C[Size*i], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, stream1));
          hipEventRecord(evento1, stream1);
          hipStreamWaitEvent(stream1, evento2);
        } else {
          hipStreamWaitEvent(stream2, evento1);
          ComplexOneStreamStep(handle, stream2, NChannels, SizeTRF, Err, U2, X2, Y2, TRF, &R[i-1]);
          hipEventRecord(evento2, stream2);

          CUDAERROMP(hipMemcpyAsync(U1, &A[Size*i], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, stream1));
          CUDAERROMP(hipMemcpyAsync(X1, &B[Size*i], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, stream1));
          CUDAERROMP(hipMemcpyAsync(Y1, &C[Size*i], sizeof(MyComplex)*Size, hipMemcpyHostToDevice, stream1));
          hipEventRecord(evento1, stream1);
          hipStreamWaitEvent(stream1, evento2);
        }
      }
      hipStreamWaitEvent(stream2, evento1);
      if ((End % 2) != 0)
        ComplexOneStreamStep(handle, stream2, NChannels, SizeTRF, Err, U2, X2, Y2, TRF, &R[End-1]);
      else
        ComplexOneStreamStep(handle, stream2, NChannels, SizeTRF, Err, U1, X1, Y1, TRF, &R[End-1]);
    }

    CUDAERROMP(hipStreamSynchronize(stream1));
    CUDAERROMP(hipStreamSynchronize(stream2));

    CUSOLVERERROMP(hipsolverDnDestroy(handle));
    CUDAERROMP(hipStreamDestroy(stream1));
    CUDAERROMP(hipStreamDestroy(stream2));
    CUDAERROMP(hipEventDestroy(evento1));
    CUDAERROMP(hipEventDestroy(evento2));

    CUDAERROMP(hipFree(TRF)); CUDAERROMP(hipFree(Err)); CUDAERROMP(hipFree(Y1)); CUDAERROMP(hipFree(U1));
    CUDAERROMP(hipFree(X1));  CUDAERROMP(hipFree(Y2));  CUDAERROMP(hipFree(U2)); CUDAERROMP(hipFree(X2));
  #ifdef OMP
  }
  #endif
  CUDAERR(hipMemcpy(Result, R, sizeof(MyComplex)*NInverses, hipMemcpyDeviceToHost));
  CUDAERR(hipFree(R)); 

  return 0;
}


int RealEventsOMP(const int NChannels, const int NInverses, const MyType *A, const MyType *B, const MyType *C, MyType *Result,  const int Cores)
{
  MyType *R=NULL;

  CUDAERR(hipMalloc((void **)&R, sizeof(MyType)*NInverses));

  #ifdef OMP
    #pragma omp parallel num_threads(Cores)
  {
  #endif
    size_t  Size;
    int     *Err=NULL, SizeTRF, i, Chunk, MyID=0, NumTh=1, Start, End;
    MyType  *U1=NULL, *X1=NULL, *Y1=NULL, *U2=NULL, *X2=NULL, *Y2=NULL, *TRF=NULL;

    hipsolverHandle_t handle;
    hipStream_t       stream1, stream2;
    hipEvent_t        evento1, evento2;

    Size=(size_t)NChannels * (size_t)NChannels;

    CUDAERROMP(hipStreamCreate(&stream1));
    CUDAERROMP(hipStreamCreate(&stream2));
    CUDAERROMP(hipEventCreate(&evento1));
    CUDAERROMP(hipEventCreate(&evento2));
    CUSOLVERERROMP(hipsolverDnCreate(&handle));
    CUSOLVERERROMP(hipsolverSetStream(handle, stream2));

    #ifdef OMP
      MyID  = omp_get_thread_num();
      NumTh = omp_get_num_threads();
    #endif
    Chunk = NInverses / NumTh;
    Start = MyID * Chunk;

    if (MyID==(NumTh-1)) { End=Start + Chunk + (NInverses % NumTh); } else { End=Start + Chunk; }

    CUDAERROMP(hipMalloc((void **)&U1, sizeof(MyType)*Size));
    CUDAERROMP(hipMalloc((void **)&X1, sizeof(MyType)*Size));
    CUDAERROMP(hipMalloc((void **)&Y1, sizeof(MyType)*Size));
    CUDAERROMP(hipMalloc((void **)&U2, sizeof(MyType)*Size));
    CUDAERROMP(hipMalloc((void **)&X2, sizeof(MyType)*Size));
    CUDAERROMP(hipMalloc((void **)&Y2, sizeof(MyType)*Size));

    CUDAERROMP(hipMemcpy(U1, &A[Start*Size], sizeof(MyType)*Size, hipMemcpyHostToDevice));
    CUDAERROMP(hipMemcpy(X1, &B[Start*Size], sizeof(MyType)*Size, hipMemcpyHostToDevice));
    CUDAERROMP(hipMemcpy(Y1, &C[Start*Size], sizeof(MyType)*Size, hipMemcpyHostToDevice));

    #ifdef SIMPLE
      CUSOLVERERROMP(hipsolverDnSpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, NChannels, U1, NChannels, &SizeTRF));
    #else
      CUSOLVERERROMP(hipsolverDnDpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_UPPER, NChannels, U1, NChannels, &SizeTRF));
    #endif
    CUDAERROMP(hipMalloc((void **)&TRF, sizeof(MyType)*SizeTRF));
    CUDAERROMP(hipMalloc((void **)&Err, sizeof(int)));

    hipEventRecord(evento1, stream1);
    if ((Start % 2) == 0)
    {
      for(i=(Start+1); i<End; i++)
      {
        if ((i % 2) != 0)
        {
          hipStreamWaitEvent(stream2, evento1);
          RealOneStreamStep(handle, stream2, NChannels, SizeTRF, Err, U1, X1, Y1, TRF, &R[i-1]);
          hipEventRecord(evento2, stream2);

          CUDAERROMP(hipMemcpyAsync(U2, &A[Size*i], sizeof(MyType)*Size, hipMemcpyHostToDevice, stream1));
          CUDAERROMP(hipMemcpyAsync(X2, &B[Size*i], sizeof(MyType)*Size, hipMemcpyHostToDevice, stream1));
          CUDAERROMP(hipMemcpyAsync(Y2, &C[Size*i], sizeof(MyType)*Size, hipMemcpyHostToDevice, stream1));
          hipEventRecord(evento1, stream1);
          hipStreamWaitEvent(stream1, evento2);
        } else {
          hipStreamWaitEvent(stream2, evento1);
          RealOneStreamStep(handle, stream2, NChannels, SizeTRF, Err, U2, X2, Y2, TRF, &R[i-1]);
          hipEventRecord(evento2, stream2);

          CUDAERROMP(hipMemcpyAsync(U1, &A[Size*i], sizeof(MyType)*Size, hipMemcpyHostToDevice, stream1));
          CUDAERROMP(hipMemcpyAsync(X1, &B[Size*i], sizeof(MyType)*Size, hipMemcpyHostToDevice, stream1));
          CUDAERROMP(hipMemcpyAsync(Y1, &C[Size*i], sizeof(MyType)*Size, hipMemcpyHostToDevice, stream1));
          hipEventRecord(evento1, stream1);
          hipStreamWaitEvent(stream1, evento2);
        }
      }
      hipStreamWaitEvent(stream2, evento1);
      if ((End % 2) == 0)
        RealOneStreamStep(handle, stream2, NChannels, SizeTRF, Err, U2, X2, Y2, TRF, &R[End-1]);
      else
        RealOneStreamStep(handle, stream2, NChannels, SizeTRF, Err, U1, X1, Y1, TRF, &R[End-1]);
    } else {
      for(i=(Start+1); i<End; i++)
      {
        if ((i % 2) == 0)
        {
          hipStreamWaitEvent(stream2, evento1);
          RealOneStreamStep(handle, stream2, NChannels, SizeTRF, Err, U1, X1, Y1, TRF, &R[i-1]);
          hipEventRecord(evento2, stream2);

          CUDAERROMP(hipMemcpyAsync(U2, &A[Size*i], sizeof(MyType)*Size, hipMemcpyHostToDevice, stream1));
          CUDAERROMP(hipMemcpyAsync(X2, &B[Size*i], sizeof(MyType)*Size, hipMemcpyHostToDevice, stream1));
          CUDAERROMP(hipMemcpyAsync(Y2, &C[Size*i], sizeof(MyType)*Size, hipMemcpyHostToDevice, stream1));
          hipEventRecord(evento1, stream1);
          hipStreamWaitEvent(stream1, evento2);
        } else {
          hipStreamWaitEvent(stream2, evento1);
          RealOneStreamStep(handle, stream2, NChannels, SizeTRF, Err, U2, X2, Y2, TRF, &R[i-1]);
          hipEventRecord(evento2, stream2);

          CUDAERROMP(hipMemcpyAsync(U1, &A[Size*i], sizeof(MyType)*Size, hipMemcpyHostToDevice, stream1));
          CUDAERROMP(hipMemcpyAsync(X1, &B[Size*i], sizeof(MyType)*Size, hipMemcpyHostToDevice, stream1));
          CUDAERROMP(hipMemcpyAsync(Y1, &C[Size*i], sizeof(MyType)*Size, hipMemcpyHostToDevice, stream1));
          hipEventRecord(evento1, stream1);
          hipStreamWaitEvent(stream1, evento2);
        }
      }
      hipStreamWaitEvent(stream2, evento1);
      if ((End % 2) != 0)
        RealOneStreamStep(handle, stream2, NChannels, SizeTRF, Err, U2, X2, Y2, TRF, &R[End-1]);
      else
        RealOneStreamStep(handle, stream2, NChannels, SizeTRF, Err, U1, X1, Y1, TRF, &R[End-1]);
    }

    CUDAERROMP(hipStreamSynchronize(stream1));
    CUDAERROMP(hipStreamSynchronize(stream2));

    CUSOLVERERROMP(hipsolverDnDestroy(handle));
    CUDAERROMP(hipStreamDestroy(stream1));
    CUDAERROMP(hipStreamDestroy(stream2));
    CUDAERROMP(hipEventDestroy(evento1));
    CUDAERROMP(hipEventDestroy(evento2));

    CUDAERROMP(hipFree(TRF)); CUDAERROMP(hipFree(Err)); CUDAERROMP(hipFree(Y1)); CUDAERROMP(hipFree(U1));
    CUDAERROMP(hipFree(X1));  CUDAERROMP(hipFree(Y2));  CUDAERROMP(hipFree(U2)); CUDAERROMP(hipFree(X2));
  #ifdef OMP
  }
  #endif
  CUDAERR(hipMemcpy(Result, R, sizeof(MyType)*NInverses, hipMemcpyDeviceToHost));
  CUDAERR(hipFree(R)); 

  return 0;
}
